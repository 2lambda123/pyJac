#include "hip/hip_runtime.h"
/* wrapper to translate to cuda arrays */

#include "header.h"
#include "gpu_memory.cuh"
#include "launch_bounds.cuh"
#include "chem_utils.cuh"
#include "rates.cuh"
#include "jacob.cuh"

#ifndef SHARED_SIZE
	#define SHARED_SIZE (0)
#endif

#define T_ID (threadIdx.x + (blockDim.x * blockIdx.x))

__global__
void k_eval_conc(const int num, const double* T, const double* pres, 
	const double* dMass, size_t pitch1, 
	const double* dMw,
	const double* dRho,
	const double* dConc, size_t pitch2);
{
	if (T_ID < num)
	{
		double mass_local[NSP];
		#pragma unroll
		for (int i = 0; i < NSP; ++i)
		{
			mass_local[i] = *((double*)((char*)dMass + i * pitch1) + T_ID);
		}

		double T_local = T[T_ID]
		double pres_local = pres[T_ID];
		double mw_avg = 0;
		double rho = 0;
		double conc_local[NSP] = {0};

		eval_conc(T_local, pres_local, mass_local, &mw_avg, &rho, conc_local);

		dMw[T_ID] = mw_avg;
		dRho[T_ID] = rho;
		#pragma unroll
		for (int i = 0; i < FWD_RATES; ++i)
		{
			*((double*)((char*)dConc + i * pitch2) + T_ID) = conc_local[i];
		}
	}
}

void cu_eval_conc (const int num, const double * T, const double * pres, const double * mass_frac, double * mw_avg, double * rho, double * conc) {
	int grid_num = num / TARGET_BLOCK_SIZE;
	//allocate device memory
	double* dT;
	double* dPres;
	double* dMass;
	double* dMw;
	double* dRho = 0;
	double* dC = 0;
	size_t pitch1 = 0, pitch2 = 0, pitch3 = 0, pitch4 = 0, pitch5 = 0, pitch6 = 0;
	cudaErrorCheck( hipMallocPitch((void**)&dT, &pitch1, num * sizeof(double), 1) );
	cudaErrorCheck( hipMallocPitch((void**)&dPres, &pitch2, num * sizeof(double), 1) );
	cudaErrorCheck( hipMallocPitch((void**)&dMass, &pitch3, num * sizeof(double), NSP) );
	cudaErrorCheck( hipMallocPitch((void**)&dMw, &pitch4, num * sizeof(double), 1) );
	cudaErrorCheck( hipMallocPitch((void**)&dRho, &pitch5, num * sizeof(double), 1) );
	cudaErrorCheck( hipMallocPitch((void**)&dC, &pitch6, num * sizeof(double), NSP) );

	//copy over
	cudaErrorCheck( hipMemcpy2D(dT, pitch1, T, num, num * sizeof(double), 1, hipMemcpyHostToDevice) );
	cudaErrorCheck( hipMemcpy2D(dPres, pitch2, pres, num, num * sizeof(double), 1, hipMemcpyHostToDevice) );
	cudaErrorCheck( hipMemcpy2D(dMass, pitch3, mass_frac, num, num * sizeof(double), NSP, hipMemcpyHostToDevice) );

	//run
	k_eval_conc<<<grid_num, TARGET_BLOCK_SIZE, SHARED_SIZE>>>(num, T, pres, dMass, pitch3, dMw, dRho, dConc, pitch6);
	//copy back
	cudaErrorCheck( hipMemcpy2D(mw_avg, num, dMw, pitch4, num * sizeof(double), 1, hipMemcpyDeviceToHost) );
	cudaErrorCheck( hipMemcpy2D(rho, num, dRho, pitch5, num * sizeof(double), 1, hipMemcpyDeviceToHost) );
	cudaErrorCheck( hipMemcpy2D(conc, num, dC, pitch6, num * sizeof(double), NSP, hipMemcpyDeviceToHost) );

	cudaErrorCheck( hipFree(dT) );
	cudaErrorCheck( hipFree(dPres) );
	cudaErrorCheck( hipFree(dMass) );
	cudaErrorCheck( hipFree(dMw) );
	cudaErrorCheck( hipFree(dRho) );
	cudaErrorCheck( hipFree(dC) );
}


__global__
void k_eval_rxn_rates(const int num, const double T, const double pres, const double * C,
	size_t pitch1, double * fwd_rxn_rates, size_t pitch2, double * rev_rxn_rates,
	size_t pitch3)
{
	if (T_ID < num) {
		double conc_local[NSP];
		#pragma unroll
		for (int i = 0; i < NSP; ++i)
		{
			conc_local[i] = *((double*)((char*)C + i * pitch1) + T_ID);
		}

		double fwd_local[FWD_RATES];

		#if REV_RATES == 0
			double* rev = 0;
		#else
			double rev_local[REV_RATES];
		#endif

		eval_rxn_rates(T, pres, conc_local, fwd_local, rev_local);

		#pragma unroll
		for (int i = 0; i < FWD_RATES; ++i)
		{
			*((double*)((char*)fwd_rxn_rates + i * pitch2) + T_ID) = fwd_local[i];
		}

		#if REV_RATES != 0
			#pragma unroll
			for (int i = 0; i < REV_RATES; ++i)
			{
				*((double*)((char*)rev_rxn_rates + i * pitch3) + T_ID) = rev_local[i];
			}
		#endif
	} 
}

void cu_eval_rxn_rates (const int num, const double T, const double pres, const double * C, double * fwd_rxn_rates, double * rev_rxn_rates) {
	int grid_num = num / TARGET_BLOCK_SIZE;
	//allocate device memory
	double* dC;
	double* dFwd;
	double* dRev = 0;
	size_t pitch1 = 0, pitch2 = 0, pitch3 = 0;
	cudaErrorCheck( hipMallocPitch((void**)&dC, &pitch1, num * sizeof(double), NSP) );
	cudaErrorCheck( hipMallocPitch((void**)&dFwd, &pitch2, num * sizeof(double), FWD_RATES) );
	#if REV_RATES != 0
		cudaErrorCheck( hipMallocPitch((void**)&dRev, &pitch3, num * sizeof(double), REV_RATES) );
	#endif

	//copy over
	cudaErrorCheck( hipMemcpy2D(dC, pitch1, C, num, num * sizeof(double), NSP, hipMemcpyHostToDevice) );

	//run
	k_eval_rxn_rates<<<grid_num, TARGET_BLOCK_SIZE, SHARED_SIZE>>>(T, pres, C, pitch1, dFwd, pitch2, dRev, pitch3);

	//copy back
	cudaErrorCheck( hipMemcpy2D(fwd_rxn_rates, num, dFwd, pitch2, num * sizeof(double), FWD_RATES, hipMemcpyDeviceToHost) );

	#if REV_RATES != 0
		cudaErrorCheck( hipMemcpy2D(rev_rxn_rates, num, dRev, pitch3, num * sizeof(double), REV_RATES, hipMemcpyDeviceToHost) );
	#endif

	cudaErrorCheck( hipFree(dC) );
	cudaErrorCheck( hipFree(dFwd) );
	cudaErrorCheck( hipFree(dRev) );
}

__global__
void k_get_rxn_pres_mod(const int num, const double T, const double pres, const double * C, size_t pitch1, double * pres_mod,
	size_t pitch2)
{
	if (T_ID < num)
	{
		double conc_local[NSP];
		double pres_mod_local[PRES_MOD_RATES];
		#pragma unroll
		for (int i = 0; i < NSP; ++i)
		{
			conc_local[i] = *((double*)((char*)C + i * pitch1) + T_ID);
		}

		get_rxn_pres_mod(T, pres, conc_local, pres_mod_local);

		#pragma unroll
		for (int i = 0; i < PRES_MOD_RATES; ++i)
		{
			*((double*)((char*)pres_mod + i * pitch2) + T_ID) = pres_mod_local[i];
		}
	}	
}

void cu_get_rxn_pres_mod (const int num, const double T, const double pres, const double * C, double * pres_mod) {
	#if PRES_MOD_RATES != 0
		int grid_num = num / TARGET_BLOCK_SIZE;
		//allocate device memory
		double* dC;
		double* dPres;
		size_t pitch1, pitch2;
		
		cudaErrorCheck( hipMallocPitch((void**)&dC, &pitch1, num * sizeof(double), NSP) );
		cudaErrorCheck( hipMallocPitch((void**)&dPres, &pitch2, num * sizeof(double), PRES_MOD_RATES) );

		//copy over
		cudaErrorCheck( hipMemcpy2D(dC, pitch1, C, num, num * sizeof(double), NSP, hipMemcpyHostToDevice) );

		//run
		k_get_rxn_pres_mod<<<grid_num, TARGET_BLOCK_SIZE, SHARED_SIZE>>>(num, T, pres, C, fwd_rxn_rates, rev_rxn_rates);

		//copy back
		cudaErrorCheck( hipMemcpy2D(pres_mod, num, dPres, pitch2, num * sizeof(double), PRES_MOD_RATES, hipMemcpyHostToDevice) );
	
		cudaErrorCheck(hipFree(dC));
		cudaErrorCheck(hipFree(dPres));
	#endif
}


__global__
void k_eval_spec_rates(const int num, const double* fwd_rates, size_t pitch1, const double * rev_rates,
	size_t pitch2, const double* pres_mod, size_t pitch3, double* spec_rates, size_t pitch4)
{
	if (T_ID < num)
	{
		double fwd_local[FWD_RATES];
		#pragma unroll
		for (int i = 0; i < FWD_RATES; ++i)
		{
			fwd_local[i] = *((double*)((char*)fwd_rates + i * pitch1) + T_ID);
		}

		#if REV_RATES != 0
			double rev_local[REV_RATES];
			#pragma unroll
			for (int i = 0; i < REV_RATES; ++i)
			{
				rev_local[i] = *((double*)((char*)rev_rates + i * pitch2) + T_ID);
			}
		#else
			double* rev_local = 0;
		#endif

		#if PRES_MOD_RATES != 0
			double pres_mod_local[PRES_MOD_RATES];
			#pragma unroll
			for (int i = 0; i < PRES_MOD_RATES; ++i)
			{
				pres_mod_local[i] = *((double*)((char*)pres_mod + i * pitch3) + T_ID);
			}
		#else
			double* pres_mod_local = 0;
		#endif

		double spec_rates_local[NSP];
		eval_spec_rates(fwd_local, rev_local, pres_mod_local, spec_rates_locals);

		#pragma unroll
		for (int i = 0; i < NSP; ++i)
		{
			*((double*)((char*)spec_rates + i * pitch4) + T_ID) = spec_rates_local[i];
		}
	}	
}

void cu_eval_spec_rates (const int num, const double * fwd_rates, const double * rev_rates, const double * pres_mod, double * spec_rates) {
	int grid_num = num / TARGET_BLOCK_SIZE;
	//allocate device memory
	double* dFwd;
	double* dRev = 0;
	double* dPres = 0;
	double* dSpec;
	size_t pitch1, pitch2 = 0, pitch3 = 0, pitch4;
	
	cudaErrorCheck( hipMallocPitch((void**)&dFwd, &pitch1, num * sizeof(double), FWD_RATES) );
	cudaErrorCheck( hipMemcpy2D(dFwd, pitch1, fwd_rates, num, num * sizeof(double), FWD_RATES, hipMemcpyHostToDevice) );
	#if REV_RATES != 0
		cudaErrorCheck( hipMallocPitch((void**)&dRev, &pitch2, num * sizeof(double), REV_RATES) );
		cudaErrorCheck( hipMemcpy2D(dRev, pitch2, rev_rates, num, num * sizeof(double), REV_RATES, hipMemcpyHostToDevice) );
	#endif

	#if PRES_MOD_RATES != 0
		cudaErrorCheck( hipMallocPitch((void**)&dPres, &pitch3, num * sizeof(double), PRES_MOD_RATES) );
		cudaErrorCheck( hipMemcpy2D(dPres, pitch3, pres_mod, num, num * sizeof(double), PRES_MOD_RATES, hipMemcpyHostToDevice) );
	#endif

	cudaErrorCheck( hipMallocPitch((void**)&dSpec, &pitch4, num * sizeof(double), NSP) );
	//run
	k_eval_spec_rates<<<grid_num, TARGET_BLOCK_SIZE, SHARED_SIZE>>>(num, dFwd, pitch1, dRev, pitch2, dPres, pitch3, dSpec, pitch4);

	//copy back
	cudaErrorCheck( hipMemcpy2D(spec_rates, num, dSpec, pitch4, num * sizeof(double), NSP, hipMemcpyHostToDevice) );

	cudaErrorCheck(hipFree(dFwd));
	#if REV_RATES != 0
		cudaErrorCheck(hipFree(dRev));
	#endif

	#if PRES_MOD_RATES != 0
		cudaErrorCheck(hipFree(dPres));
	#endif

	cudaErrorCheck(hipFree(dSpec));
}


__global__
void k_dydt(const int num, const double pres, const double* y, size_t pitch1, double * dy,
	size_t pitch2)
{
	if (T_ID < num)
	{
		double y_local[NN];
		#pragma unroll
		for (int i = 0; i < NN; ++i)
		{
			y_local[i] = *((double*)((char*)y + i * pitch1) + T_ID);
		}

		double dy_local[NN];
		dydt(0, pres, y_local, dy_local);

		#pragma unroll
		for (int i = 0; i < NN; ++i)
		{
			*((double*)((char*)dy + i * pitch2) + T_ID) = dy_local[i];
		}
	}	
}

void cu_dydt (const int num, const double pres, const double* y, double* dy) {
	int grid_num = num / TARGET_BLOCK_SIZE;
	//allocate device memory
	double* dY;
	double* dDy;
	size_t pitch1, pitch2;
	
	cudaErrorCheck( hipMallocPitch((void**)&dY, &pitch1, num * sizeof(double), NN) );
	cudaErrorCheck( hipMemcpy2D(dY, pitch1, y, num, num * sizeof(double), NN, hipMemcpyHostToDevice) );

	cudaErrorCheck( hipMallocPitch((void**)&dDy, &pitch2, num * sizeof(double), NN) );
	//run
	k_dydt<<<grid_num, TARGET_BLOCK_SIZE, SHARED_SIZE>>>(num, dY, pitch1, dDy, pitch2);

	//copy back
	cudaErrorCheck( hipMemcpy2D(dy, num, dDy, pitch2, num * sizeof(double), NN, hipMemcpyHostToDevice) );

	cudaErrorCheck(hipFree(dY));
	cudaErrorCheck(hipFree(dDy));
}

__global__
void k_eval_jacob(const int num, const double pres, const double* y, size_t pitch1, double * jac,
	size_t pitch2)
{
	if (T_ID < num)
	{
		double y_local[NN];
		#pragma unroll
		for (int i = 0; i < NN; ++i)
		{
			y_local[i] = *((double*)((char*)y + i * pitch1) + T_ID);
		}

		double jac_local[NN * NN] = {0};
		eval_jacob(0, pres, y_local, jac_local);

		#pragma unroll
		for (int i = 0; i < NN * NN; ++i)
		{
			*((double*)((char*)jac + i * pitch2) + T_ID) = jac_local[i];
		}
	}	
}

void cu_eval_jacob (const int num, const double t, const double pres, const double* y, double* jac) {
	int grid_num = num / TARGET_BLOCK_SIZE;
	//allocate device memory
	double* dY;
	double* dJac;
	size_t pitch1, pitch2;
	
	cudaErrorCheck( hipMallocPitch((void**)&dY, &pitch1, num * sizeof(double), NN) );
	cudaErrorCheck( hipMemcpy2D(dY, pitch1, y, num, num * sizeof(double), NN, hipMemcpyHostToDevice) );

	cudaErrorCheck( hipMallocPitch((void**)&dJac, &pitch2, num * sizeof(double), NN * NN) );
	//run
	k_eval_jacob<<<grid_num, TARGET_BLOCK_SIZE, SHARED_SIZE>>>(num, dY, pitch1, dJac, pitch2);

	//copy back
	cudaErrorCheck( hipMemcpy2D(jac, num, dJac, pitch2, num * sizeof(double), NN * NN, hipMemcpyHostToDevice) );

	cudaErrorCheck(hipFree(dY));
	cudaErrorCheck(hipFree(dJac));
}